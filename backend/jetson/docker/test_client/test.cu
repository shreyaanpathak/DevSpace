
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    printf("Starting CUDA test...\n");
    fflush(stdout);  // Force print
    
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    
    if (err != hipSuccess) {
        printf("Error getting device count: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    printf("Found %d CUDA devices\n", deviceCount);
    return 0;
}
